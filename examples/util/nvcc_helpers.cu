#ifndef _CUDA_
#error "Something went wrong..."
#endif

void ispc_malloc(void **ptr, const size_t size)
{
  hipMallocManaged(ptr, size);
}
void ispc_free(void *ptr)
{
  hipFree(ptr);
}
void ispc_memset(void *ptr, int value, size_t size)
{
  hipMemset(ptr, value, size);
}
void ispcSetMallocHeapLimit(size_t value)
{
  hipDeviceSetLimit(hipLimitMallocHeapSize,value);
}
void ispcSetStackLimit(size_t value)
{
  hipDeviceSetLimit(hipLimitStackSize,value);
}
unsigned long long ispcGetMallocHeapLimit()
{
  size_t value;
  hipDeviceGetLimit(&value, hipLimitMallocHeapSize);
  return value;
}
unsigned long long ispcGetStackLimit()
{
  size_t value;
  hipDeviceGetLimit(&value, hipLimitStackSize);
  return value;
}
void * ispcMemcpy(void *dest,  void *src,  size_t num)
{
  hipMemcpy(dest, src, num, hipMemcpyDefault);
  return dest;
}


