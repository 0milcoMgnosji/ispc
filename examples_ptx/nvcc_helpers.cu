#ifndef _CUDA_
#error "Something went wrong..."
#endif

void ispc_malloc(void **ptr, const size_t size)
{
  hipMallocManaged(ptr, size);
}
void ispc_free(void *ptr)
{
  hipFree(ptr);
}
void ispc_memset(void *ptr, int value, size_t size)
{
  hipMemset(ptr, value, size);
}


