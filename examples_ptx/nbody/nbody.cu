#include "hip/hip_runtime.h"
typedef double real;


static uniform real * uniform accx = NULL;
static uniform real * uniform accy;
static uniform real * uniform accz;
static uniform real * uniform gpotList;

export
void openNbody(const uniform int n)
{
  assert(accx == NULL);
  accx = uniform new uniform real[n];
  accy = uniform new uniform real[n];
  accz = uniform new uniform real[n];
  gpotList = uniform new uniform real[n];
}

export 
void closeNbody()
{
  assert(accx != NULL);
  delete accx;
  delete accy;
  delete accz;
  delete gpotList;
}


task 
void computeForces(
    uniform int  nbodies,
    uniform real posx[],
    uniform real posy[],
    uniform real posz[],
    uniform real mass[])
{
  const uniform int blockIdx = taskIndex;
  const uniform int blockDim = (nbodies + taskCount - 1)/taskCount;
  const uniform int blockBeg =     blockIdx * blockDim;
  const uniform int blockEnd = min(blockBeg + blockDim, nbodies);

#if 0
  uniform real gpotLoc = 0;
  for (uniform int i = blockBeg; i < blockEnd; i++)
  {
    const real iposx = posx[i];
    const real iposy = posy[i];
    const real iposz = posz[i];
    real iaccx = 0;
    real iaccy = 0;
    real iaccz = 0;
    real igpot = 0;
    foreach (j = 0 ... nbodies)
    {
      const real jposx = posx[j];
      const real jposy = posy[j];
      const real jposz = posz[j];
      const real jmass = mass[j];
      const real    dx  = jposx - iposx;
      const real    dy  = jposy - iposy;
      const real    dz  = jposz - iposz;
      const real    r2  = dx*dx + dy*dy + dz*dz;
      const real  rinv  = r2 > 0.0d ? rsqrt((float)r2) : 0;
      const real mrinv  = -jmass * rinv;
      const real mrinv3 = mrinv * rinv*rinv;

      iaccx += mrinv3 * dx;
      iaccy += mrinv3 * dy;
      iaccz += mrinv3 * dz;
      igpot += mrinv;
    }
    accx[i]  = reduce_add(iaccx);
    accy[i]  = reduce_add(iaccy);
    accz[i]  = reduce_add(iaccz);
    gpotLoc += reduce_add(igpot);
  }
  gpotList[taskIndex] = gpotLoc;
#else
  real gpotLoc = 0;
  foreach (i = blockBeg ... blockEnd)
  {
    const real iposx = posx[i];
    const real iposy = posy[i];
    const real iposz = posz[i];
    real iaccx = 0;
    real iaccy = 0;
    real iaccz = 0;
    real igpot = 0;
    for (uniform int j = 0; j < nbodies; j += 1)
    {
#define STEP(jk) {\
      const real jposx = posx[j+jk]; \
      const real jposy = posy[j+jk]; \
      const real jposz = posz[j+jk]; \
      const real jmass = mass[j+jk]; \
      const real    dx  = jposx - iposx; \
      const real    dy  = jposy - iposy; \
      const real    dz  = jposz - iposz; \
      const real    r2  = dx*dx + dy*dy + dz*dz; \
      const real  rinv  = r2 > 0.0d ? rsqrt((float)r2) : 0; \
      const real mrinv  = -jmass * rinv; \
      const real mrinv3 = mrinv * rinv*rinv; \
 \
      iaccx += mrinv3 * dx; \
      iaccy += mrinv3 * dy; \
      iaccz += mrinv3 * dz; \
      igpot += mrinv; \
}
    STEP(0)
    }
    accx[i]  = iaccx;
    accy[i]  = iaccy;
    accz[i]  = iaccz;
    gpotLoc += igpot;
  }
  gpotList[taskIndex] = reduce_add(gpotLoc);
#endif
}

task
void updatePositions(
    uniform int  nbodies,
    uniform real posx[],
    uniform real posy[],
    uniform real posz[],
    uniform real velx[],
    uniform real vely[],
    uniform real velz[],
    uniform real dt)
{
  const uniform int blockIdx = taskIndex;
  const uniform int blockDim = (nbodies + taskCount - 1)/taskCount;
  const uniform int blockBeg =     blockIdx * blockDim;
  const uniform int blockEnd = min(blockBeg + blockDim, nbodies);

  foreach (i = blockBeg ... blockEnd)
  {
    posx[i] += dt*velx[i];
    posy[i] += dt*vely[i];
    posz[i] += dt*velz[i];
  }
}

task
void updateVelocities(
    uniform int  nbodies,
    uniform real velx[],
    uniform real vely[],
    uniform real velz[],
    uniform real dt)
{
  const uniform int blockIdx = taskIndex;
  const uniform int blockDim = (nbodies + taskCount - 1)/taskCount;
  const uniform int blockBeg =     blockIdx * blockDim;
  const uniform int blockEnd = min(blockBeg + blockDim, nbodies);

  foreach (i = blockBeg ... blockEnd)
  {
    velx[i] += dt*accx[i];
    vely[i] += dt*accy[i];
    velz[i] += dt*accz[i];
  }
}

export
void nbodyIntegrate(
    uniform int  nSteps,
    uniform int  nbodies,
    uniform real dt,
    uniform real posx[],
    uniform real posy[],
    uniform real posz[],
    uniform real mass[],
    uniform real velx[],
    uniform real vely[],
    uniform real velz[],
    uniform real energies[])
{
  uniform int nTasks = num_cores()*4;
#ifdef __NVPTX__
  nTasks = nbodies/(4*programCount);
#endif
  assert((nbodies % nTasks) == 0);

  for (uniform int step = 0; step < nSteps; step++)
  { 
    launch [nTasks] updatePositions(nbodies, posx, posy, posz, velx, vely, velz,dt);
    sync;
    launch [nTasks] computeForces(nbodies, posx, posy, posz, mass);
    sync;
    launch [nTasks] updateVelocities(nbodies, posx, posy, posz, dt);
    sync;
  }

  if (energies != NULL)
  {
    real gpotLoc = 0;
    foreach (i = 0 ... nTasks)
      gpotLoc += gpotList[i];
    energies[0] = reduce_add(gpotLoc);
  }
}


